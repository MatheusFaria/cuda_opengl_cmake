#include "hip/hip_runtime.h"
#include "cudahelperlib.h"

#include <cstdio>

__host__ void cudaErrorCheck(hipError_t err)
{
    if (err != hipSuccess)
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
}

__device__ float4 pickRandomFloat4(hiprandState * randState)
{
    return make_float4(hiprand_uniform(randState),
                       hiprand_uniform(randState),
                       hiprand_uniform(randState), 1);
}
